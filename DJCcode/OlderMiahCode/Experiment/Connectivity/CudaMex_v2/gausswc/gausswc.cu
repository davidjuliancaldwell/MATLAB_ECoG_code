#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <mex.h>

#define MEX_CHECK_RETURN(in) {hipError_t __errLocal = in; if(__errLocal != hipSuccess) { mexPrintf("ERROR: file %s, line %i\n  CUDA call \n\t" #in " \n  returned error (%i) - \"%s\"\n", __FILE__, __LINE__, __errLocal, hipGetErrorString(__errLocal)); } }

__device__ __constant__ int rvGVectorLengths;
__device__ __constant__ int rvGWindowSize;
__device__ __constant__ int rvGPrePostLag;
__device__ __constant__ int rvGLeftAlpha;
__device__ __constant__ int rvGRightAlpha;
__device__ __constant__ int rvGNumElsX;
__device__ __constant__ int rvGNumElsY;
__device__ __constant__ int rvGSharedMemorySizeX;
__device__ __constant__ int rvGSharedMemorySizeY;
__device__ __constant__ float rvGWindowVals[1024];

__device__ float kernel_cov(float *pA, float *pB, int pElements) {

	float summation[2] = {0.0f,0.0f};
	//Calculate the means for each vector
	for(int i = 0; i < pElements; i++) {
		summation[0] = summation[0] + pA[i];
		summation[1] = summation[1] + pB[i];
	}
	summation[0] = summation[0] / pElements;
	summation[1] = summation[1] / pElements;

	float covMeasure = 0.0f;
	//Calculate the means for each vector
	for(int i = 0; i < pElements; i++) {
		covMeasure = covMeasure + ((pA[i]*rvGWindowVals[i])-summation[0])*(pB[i]-summation[1]);
		//covMeasure = covMeasure + ((pA[i])-summation[0])*(pB[i]-summation[1]);
	}
	covMeasure = covMeasure / (pElements-1);
	return covMeasure;
	
	//printf("Means of each vector:\n  a=%f\n  b=%f\nCovariance: %f\n", summation[0], summation[1], covMeasure);
}



__global__ void windowed_cov(float *pX, float *pY, float *pOut, int pRowOffset) {

	extern __shared__ float sharedMem[];

	int t = blockIdx.y * gridDim.x + blockIdx.x;
	//t = t + 1;

	int yRange[2];
	yRange[0] = t-rvGLeftAlpha;
	yRange[1] = t+rvGRightAlpha;
	int xRange[2];
	xRange[0] = t - rvGPrePostLag - rvGLeftAlpha;
	xRange[1] = t + rvGPrePostLag + rvGRightAlpha;

	int outputRow = threadIdx.x + pRowOffset;

	int outputLocation = t * (2*rvGPrePostLag + 1) + outputRow;
	
	
	/*if(xRange[0] < 0 || yRange[0] < 0 || xRange[1] > rvGVectorLengths || yRange[1] > rvGVectorLengths) {
		pOut[outputLocation] = 0;
		return;
	}*/

	float *ySharedMem = (float*) sharedMem;
	float *xSharedMem = (float*) &ySharedMem[rvGNumElsY];
	
	//__shared__ float ySharedMem[rvGSharedMemorySizeY];
	//__shared__ float xSharedMem[rvGSharedMemorySizeX];

	/* 
		Load y into shared memory
	*/
	int globalMemLocation;



	//Pull the Y into shared memory
	int localMemPos = threadIdx.x;
	while(localMemPos < rvGWindowSize+1) {
		globalMemLocation = min(max(yRange[0] + localMemPos,0),rvGVectorLengths-1);
		ySharedMem[localMemPos] = pY[globalMemLocation];
		localMemPos = localMemPos + blockDim.x;
	}

	__syncthreads();
	// Y is loaded!

	/*if(outputRow < 601) {
		pOut[outputLocation] = ySharedMem[300];
	}
	return;*/

	/* 
		Load x into shared memory
	*/

	// Total evaluation range for this sample t
	int xEvalRange[2];
	xEvalRange[0] = t - rvGPrePostLag;
	xEvalRange[1] = t + rvGPrePostLag;


	// Evaluation range for this current kernel loop
	int kernelLoopEvalRange[2];
	kernelLoopEvalRange[0] = xEvalRange[0] + pRowOffset;
	kernelLoopEvalRange[1] = min(kernelLoopEvalRange[0] + blockDim.x,xEvalRange[1]);

	

	int blockMemoryRange[2];
	blockMemoryRange[0] = kernelLoopEvalRange[0] - rvGLeftAlpha;
	blockMemoryRange[1] = min(kernelLoopEvalRange[1] + rvGRightAlpha, xEvalRange[1] + rvGRightAlpha);


	localMemPos = threadIdx.x;
	
	while(localMemPos < blockMemoryRange[1] - blockMemoryRange[0]+1) {
		globalMemLocation = min(max(blockMemoryRange[0] + localMemPos,0), rvGVectorLengths-1);

		xSharedMem[localMemPos] = pX[globalMemLocation];
		localMemPos = localMemPos + blockDim.x;
	}

	__syncthreads();

	if(outputRow >= (2 * rvGPrePostLag + 1)) {
		return;
	}
	

	float cov = kernel_cov(ySharedMem, xSharedMem + threadIdx.x, rvGWindowSize + 1);
	
	pOut[outputLocation] = cov;
	//pOut[outputLocation] = blockMemoryRange[1];
}

void RunCuda(float *pStaticVector, float *pMovingVector, float *pOutput, int pNumElements, int pWindowSize, int pPrePostLag,float *pWindow) {

	int deviceCount;
	MEX_CHECK_RETURN(hipGetDeviceCount(&deviceCount));

	if(deviceCount == 2) {
		hipSetDevice(1);
	}

	// Calculate the size of the output
	int heightOut = pPrePostLag * 2 + 1;
	int widthOut = pNumElements;
	int outputLength = heightOut * widthOut;

	int leftAlpha = ceil(pWindowSize/2.0f);
	int rightAlpha = floor(pWindowSize/2.0f);
	int numElsX = pWindowSize + 2 * pPrePostLag + 1;
	int numElsY = pWindowSize;
	int smXSz = numElsX*sizeof(float);
	int smYSz = numElsY*sizeof(float);

	//THIS WORKS
	int kernelRunsPerSample = 1;
	int threadsPerBlock = ceil(heightOut / 32.0f) * 32;

	while (threadsPerBlock > 512) {
		kernelRunsPerSample ++;
		threadsPerBlock = ceil((heightOut / kernelRunsPerSample)/32.0f)*32;
	}

	//Need to setup the number of blocks horizontally

	int verticalBlocksInGrid = 1;
	int horizontalBlocksInGrid = int(pNumElements * 1.0f / verticalBlocksInGrid); 

	while(horizontalBlocksInGrid > 65535) {
		verticalBlocksInGrid++;
		horizontalBlocksInGrid = int(pNumElements * 1.0f / verticalBlocksInGrid); 
	}



	//Now allocate the device data;

	float *deviceX, *deviceY, *deviceOutput;
	//mexPrintf("Allocating device input memory of length %i\n",pNumElements);
	MEX_CHECK_RETURN(hipMalloc(&deviceX, sizeof(float) * pNumElements)); 
	MEX_CHECK_RETURN(hipMalloc(&deviceY, sizeof(float) * pNumElements));

	//mexPrintf("Allocating device output (%ix%i) memory of length %.2fMB (%i bytes)\n",heightOut, widthOut, outputLength/1024.0f/1024.0f*sizeof(float),outputLength*sizeof(float));
	MEX_CHECK_RETURN(hipMalloc(&deviceOutput, sizeof(float) * outputLength));

	//mexPrintf("Setting constants\n");
	MEX_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL("rvGVectorLengths"), &pNumElements,sizeof(int)));
	MEX_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL("rvGWindowSize"), &pWindowSize,sizeof(int)));
	MEX_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL("rvGPrePostLag"), &pPrePostLag,sizeof(int)));
	MEX_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL("rvGLeftAlpha"), &leftAlpha,sizeof(int)));
	MEX_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL("rvGRightAlpha"), &rightAlpha,sizeof(int)));
	MEX_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL("rvGNumElsX"), &numElsX, sizeof(int)));
	MEX_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL("rvGNumElsY"), &numElsY, sizeof(int)));
	MEX_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL("rvGSharedMemorySizeX"), &smXSz, sizeof(int)));
	MEX_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL("rvGSharedMemorySizeY"), &smYSz, sizeof(int)));
	MEX_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL("rvGWindowVals"), pWindow, sizeof(float)*pWindowSize));

	//mexPrintf("Copying local memory to device\n");
	MEX_CHECK_RETURN(hipMemcpy( deviceX, pStaticVector, sizeof(float) * pNumElements, hipMemcpyHostToDevice ));
	MEX_CHECK_RETURN(hipMemcpy( deviceY, pMovingVector, sizeof(float) * pNumElements, hipMemcpyHostToDevice ));

	//Don't think we need to do this
	//MEX_CHECK_RETURN(hipMemcpy( deviceOutput, pOutput, sizeof(float) * outputLength, hipMemcpyHostToDevice ));



	dim3 dimGrid(horizontalBlocksInGrid, verticalBlocksInGrid);
	dim3 dimBlock(threadsPerBlock);

	//kernelRunsPerSample = 1; //mexPrintf("**HARDCODED TO kernelRunsPerSample=%i**\n",kernelRunsPerSample);

	//mexPrintf("Starting CUDA!! will run kernel %ix times with Grid dimension %ix%i, TBP(%i)\n", kernelRunsPerSample, horizontalBlocksInGrid, verticalBlocksInGrid, threadsPerBlock);
	for(int runNum = 0; runNum < kernelRunsPerSample; runNum++) {
		windowed_cov<<<dimGrid, dimBlock,smXSz + smYSz>>>(deviceX, deviceY, deviceOutput, runNum * threadsPerBlock);
		MEX_CHECK_RETURN(hipDeviceSynchronize());
		MEX_CHECK_RETURN(hipGetLastError());
	}


	//mexPrintf("Pulling results from GPU\n");
	MEX_CHECK_RETURN(hipMemcpy( pOutput, deviceOutput, sizeof(float) * outputLength, hipMemcpyDeviceToHost ));

   	//mexPrintf("Freeing device output memory\n");
	MEX_CHECK_RETURN(hipFree(deviceOutput));

	//mexPrintf("Freeing device memory\n");
	MEX_CHECK_RETURN(hipFree(deviceX));
	MEX_CHECK_RETURN(hipFree(deviceY));
	
	return;
}
