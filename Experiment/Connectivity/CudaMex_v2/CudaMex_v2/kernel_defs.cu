#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <mex.h>

#define MEX_CHECK_RETURN(in) {hipError_t __errLocal = in; if(__errLocal != hipSuccess) { mexPrintf("ERROR: file %s, line %i\n  CUDA call \n\t" #in " \n  returned error (%i) - \"%s\"\n", __FILE__, __LINE__, __errLocal, hipGetErrorString(__errLocal)); } }

__device__ __constant__ int rvVectorLengths;
__device__ __constant__ int rvWindowSize;
__device__ __constant__ int rvPrePostLag;
__device__ __constant__ int rvLeftAlpha;
__device__ __constant__ int rvRightAlpha;
__device__ __constant__ int rvNumElsX;
__device__ __constant__ int rvNumElsY;
__device__ __constant__ int rvSharedMemorySizeX;
__device__ __constant__ int rvSharedMemorySizeY;

__device__ float kernel_cov(float *pA, float *pB, int pElements) {
	float summation[2] = {0.0f,0.0f};
	//Calculate the means for each vector
	for(int i = 0; i < pElements; i++) {
		summation[0] = summation[0] + pA[i];
		summation[1] = summation[1] + pB[i];
	}
	summation[0] = summation[0] / pElements;
	summation[1] = summation[1] / pElements;

	float covMeasure = 0.0f;
	//Calculate the means for each vector
	for(int i = 0; i < pElements; i++) {
		covMeasure = covMeasure + (pA[i]-summation[0])*(pB[i]-summation[1]);
	}
	covMeasure = covMeasure / (pElements-1);
	return covMeasure;
}



__global__ void windowed_cov(float *pX, float *pY, float *pOut, int pRowOffset) {

	extern __shared__ float sharedMem[];

	int t = blockIdx.y * gridDim.x + blockIdx.x;
	//t = t + 1;

	int yRange[2];
	yRange[0] = t-rvLeftAlpha;
	yRange[1] = t+rvRightAlpha;
	int xRange[2];
	xRange[0] = t - rvPrePostLag - rvLeftAlpha;
	xRange[1] = t + rvPrePostLag + rvRightAlpha;

	int outputRow = threadIdx.x + pRowOffset;

	int outputLocation = t * (2*rvPrePostLag + 1) + outputRow;
	
	
	/*if(xRange[0] < 0 || yRange[0] < 0 || xRange[1] > rvVectorLengths || yRange[1] > rvVectorLengths) {
		pOut[outputLocation] = 0;
		return;
	}*/

	float *ySharedMem = (float*) sharedMem;
	float *xSharedMem = (float*) &ySharedMem[rvNumElsY];
	
	//__shared__ float ySharedMem[rvSharedMemorySizeY];
	//__shared__ float xSharedMem[rvSharedMemorySizeX];

	/* 
		Load y into shared memory
	*/
	int globalMemLocation;



	//Pull the Y into shared memory
	int localMemPos = threadIdx.x;
	while(localMemPos < rvWindowSize+1) {
		globalMemLocation = min(max(yRange[0] + localMemPos,0),rvVectorLengths-1);
		ySharedMem[localMemPos] = pY[globalMemLocation];
		localMemPos = localMemPos + blockDim.x;
	}

	__syncthreads();
	// Y is loaded!

	/*if(outputRow < 601) {
		pOut[outputLocation] = ySharedMem[300];
	}
	return;*/

	/* 
		Load x into shared memory
	*/

	// Total evaluation range for this sample t
	int xEvalRange[2];
	xEvalRange[0] = t - rvPrePostLag;
	xEvalRange[1] = t + rvPrePostLag;


	// Evaluation range for this current kernel loop
	int kernelLoopEvalRange[2];
	kernelLoopEvalRange[0] = xEvalRange[0] + pRowOffset;
	kernelLoopEvalRange[1] = min(kernelLoopEvalRange[0] + blockDim.x,xEvalRange[1]);

	

	int blockMemoryRange[2];
	blockMemoryRange[0] = kernelLoopEvalRange[0] - rvLeftAlpha;
	blockMemoryRange[1] = min(kernelLoopEvalRange[1] + rvRightAlpha, xEvalRange[1] + rvRightAlpha);


	localMemPos = threadIdx.x;
	
	while(localMemPos < blockMemoryRange[1] - blockMemoryRange[0]+1) {
		globalMemLocation = min(max(blockMemoryRange[0] + localMemPos,0), rvVectorLengths-1);

		xSharedMem[localMemPos] = pX[globalMemLocation];
		localMemPos = localMemPos + blockDim.x;
	}

	__syncthreads();

	if(outputRow >= (2 * rvPrePostLag + 1)) {
		return;
	}
	

	float cov = kernel_cov(ySharedMem, xSharedMem + threadIdx.x, rvWindowSize + 1);
	
	pOut[outputLocation] = cov;
	//pOut[outputLocation] = blockMemoryRange[1];
}

void RunCuda(float *pStaticVector, float *pMovingVector, float *pOutput, int pNumElements, int pWindowSize, int pPrePostLag) {

	int deviceCount;
	MEX_CHECK_RETURN(hipGetDeviceCount(&deviceCount));

	if(deviceCount == 2) {
		hipSetDevice(1);
	}

	// Calculate the size of the output
	int heightOut = pPrePostLag * 2 + 1;
	int widthOut = pNumElements;
	int outputLength = heightOut * widthOut;

	int leftAlpha = ceil(pWindowSize/2.0f);
	int rightAlpha = floor(pWindowSize/2.0f);
	int numElsX = pWindowSize + 2 * pPrePostLag + 1;
	int numElsY = pWindowSize;
	int smXSz = numElsX*sizeof(float);
	int smYSz = numElsY*sizeof(float);

	//THIS WORKS
	int kernelRunsPerSample = 1;
	int threadsPerBlock = ceil(heightOut / 32.0f) * 32;

	while (threadsPerBlock > 512) {
		kernelRunsPerSample ++;
		threadsPerBlock = ceil((heightOut / kernelRunsPerSample)/32.0f)*32;
	}

	//Need to setup the number of blocks horizontally

	int verticalBlocksInGrid = 1;
	int horizontalBlocksInGrid = int(pNumElements * 1.0f / verticalBlocksInGrid); 

	while(horizontalBlocksInGrid > 65535) {
		verticalBlocksInGrid++;
		horizontalBlocksInGrid = int(pNumElements * 1.0f / verticalBlocksInGrid); 
	}



	//Now allocate the device data;

	float *deviceX, *deviceY, *deviceOutput;
	//mexPrintf("Allocating device input memory of length %i\n",pNumElements);
	MEX_CHECK_RETURN(hipMalloc(&deviceX, sizeof(float) * pNumElements)); 
	MEX_CHECK_RETURN(hipMalloc(&deviceY, sizeof(float) * pNumElements));

	//mexPrintf("Allocating device output (%ix%i) memory of length %.2fMB (%i bytes)\n",heightOut, widthOut, outputLength/1024.0f/1024.0f*sizeof(float),outputLength*sizeof(float));
	MEX_CHECK_RETURN(hipMalloc(&deviceOutput, sizeof(float) * outputLength));

	//mexPrintf("Setting constants\n");
	MEX_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL("rvVectorLengths"), &pNumElements,sizeof(int)));
	MEX_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL("rvWindowSize"), &pWindowSize,sizeof(int)));
	MEX_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL("rvPrePostLag"), &pPrePostLag,sizeof(int)));
	MEX_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL("rvLeftAlpha"), &leftAlpha,sizeof(int)));
	MEX_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL("rvRightAlpha"), &rightAlpha,sizeof(int)));
	MEX_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL("rvNumElsX"), &numElsX, sizeof(int)));
	MEX_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL("rvNumElsY"), &numElsY, sizeof(int)));
	MEX_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL("rvSharedMemorySizeX"), &smXSz, sizeof(int)));
	MEX_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL("rvSharedMemorySizeY"), &smYSz, sizeof(int)));

	//mexPrintf("Copying local memory to device\n");
	MEX_CHECK_RETURN(hipMemcpy( deviceX, pStaticVector, sizeof(float) * pNumElements, hipMemcpyHostToDevice ));
	MEX_CHECK_RETURN(hipMemcpy( deviceY, pMovingVector, sizeof(float) * pNumElements, hipMemcpyHostToDevice ));

	//Don't think we need to do this
	//MEX_CHECK_RETURN(hipMemcpy( deviceOutput, pOutput, sizeof(float) * outputLength, hipMemcpyHostToDevice ));



	dim3 dimGrid(horizontalBlocksInGrid, verticalBlocksInGrid);
	dim3 dimBlock(threadsPerBlock);

	//kernelRunsPerSample = 1; //mexPrintf("**HARDCODED TO kernelRunsPerSample=%i**\n",kernelRunsPerSample);

	//mexPrintf("Starting CUDA!! will run kernel %ix times with Grid dimension %ix%i, TBP(%i)\n", kernelRunsPerSample, horizontalBlocksInGrid, verticalBlocksInGrid, threadsPerBlock);
	for(int runNum = 0; runNum < kernelRunsPerSample; runNum++) {
		windowed_cov<<<dimGrid, dimBlock,smXSz + smYSz>>>(deviceX, deviceY, deviceOutput, runNum * threadsPerBlock);
		MEX_CHECK_RETURN(hipDeviceSynchronize());
		MEX_CHECK_RETURN(hipGetLastError());
	}


	//mexPrintf("Pulling results from GPU\n");
	MEX_CHECK_RETURN(hipMemcpy( pOutput, deviceOutput, sizeof(float) * outputLength, hipMemcpyDeviceToHost ));

   	//mexPrintf("Freeing device output memory\n");
	MEX_CHECK_RETURN(hipFree(deviceOutput));

	//mexPrintf("Freeing device memory\n");
	MEX_CHECK_RETURN(hipFree(deviceX));
	MEX_CHECK_RETURN(hipFree(deviceY));
	
	return;
}
